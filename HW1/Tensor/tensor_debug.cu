#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <cassert>
#include <cstdlib>
#include <fstream>
#include <cstring>

typedef float scalar_t;

const int kCudaThreadsNum = 512;
inline int CudaGetBlocks(const int N)
{
    return (N + kCudaThreadsNum - 1) / kCudaThreadsNum;
}
#define CUDA_KERNEL_LOOP(n) \
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

__global__ void init_gpu(scalar_t* data, int size_)
{
    CUDA_KERNEL_LOOP(size_)
    {
        data[i] = 0.0f;
    }
}

__global__ void relu_forward_gpu(scalar_t* in, scalar_t* out, int n)
{
    CUDA_KERNEL_LOOP(n)
    {
        out[i] = in[i] > 0 ? in[i] : 0;
    }
}

__global__ void relu_backward_gpu(scalar_t* out_grad, scalar_t* in, scalar_t* in_grad, int n)
{
    CUDA_KERNEL_LOOP(n)
    {
        in_grad[i] = in[i] > 0 ? out_grad[i] : 0;
    }
}

__global__ void sigmoid_forward_gpu(scalar_t* in, scalar_t* out, int n)
{
    CUDA_KERNEL_LOOP(n)
    {
        out[i] = 1.0f / (1.0f + expf(-in[i]));
    }
}

__global__ void sigmoid_backward_gpu(scalar_t* out_grad, scalar_t* in, scalar_t* in_grad, int n)
{
    CUDA_KERNEL_LOOP(n)
    {
        scalar_t out = 1.0f / (1.0f + expf(-in[i]));
        in_grad[i] = out_grad[i] * out * (1 - out);
    }
}

class Tensor
{
public:
    std::vector<int> shape;
    bool host;
    size_t size;
    scalar_t* data;
    scalar_t* allocate_cpu()
    {
        scalar_t* data_ = new scalar_t[size];
        return data_;
    }
    scalar_t* allocate_gpu()
    {
        scalar_t* data_ = nullptr;
        hipMalloc(&data_, size * sizeof(scalar_t));
        return data_;
    }
    Tensor(std::vector<int>& shape, bool host=true)
    : shape(shape), host(host), data(nullptr)
    {
        size = 1;
        for(int dim : shape)
        {
            size *= dim;
        }
        if(host)
        {
            data = allocate_cpu();
            memset(data, 0, size * sizeof(scalar_t));
        }
        else
        {
            data = allocate_gpu();
            init_gpu<<<CudaGetBlocks(size), kCudaThreadsNum>>>(data, size);
        }
    }
    ~Tensor()
    {
        if(data)
        {
            if(host)
            {
                delete[] data;
            }
            else
            {
                hipFree(data);
            }
        }
    }
    Tensor(Tensor&& other) noexcept
    : shape(other.shape), host(other.host), size(other.size), data(other.data)
    {
        other.data = nullptr;
    }
    Tensor& operator=(Tensor&& other) noexcept
    {
        if(this != &other)
        {
            if(this->data)
            {
                if(this->host)
                {
                    delete[] this->data;
                }
                else
                {
                    hipFree(this->data);
                }
            }
            this->data = other.data;
            other.data = nullptr;
            this->shape = other.shape;
            this->host = other.host;
            this->size = other.size;
        }
        return *this;
    }
    Tensor(const Tensor& other)
    : shape(other.shape), host(other.host), size(other.size)
    {
        if(this->host)
        {
            this->data = allocate_cpu();
            hipMemcpy(this->data, other.data, this->size * sizeof(scalar_t), hipMemcpyHostToHost);
        }
        else
        {
            this->data = allocate_gpu();
            hipMemcpy(this->data, other.data, this->size * sizeof(scalar_t), hipMemcpyDeviceToDevice);
        }
    }
    Tensor& operator=(const Tensor& other)
    {
        if(this != &other)
        {
            if(this->data)
            {
                if(this->host)
                {
                    delete[] this->data;
                }
                else
                {
                    hipFree(this->data);
                }
            }
            this->shape = other.shape;
            this->host = other.host;
            this->size = other.size;
            if(this->host)
            {
                this->data = allocate_cpu();
                hipMemcpy(this->data, other.data, this->size * sizeof(scalar_t), hipMemcpyHostToHost);
            }
            else
            {
                this->data = allocate_gpu();
                hipMemcpy(this->data, other.data, this->size * sizeof(scalar_t), hipMemcpyDeviceToDevice);
            }
        }
        return *this;
    }
    Tensor cpu()
    {
        Tensor tensor(this->shape, true);
        if(this->host)
        {
            hipMemcpy(tensor.data, this->data, this->size * sizeof(scalar_t), hipMemcpyHostToHost);
        }
        else
        {
            hipMemcpy(tensor.data, this->data, this->size * sizeof(scalar_t), hipMemcpyDeviceToHost);
        }
        return tensor;
    }
    Tensor gpu()
    {
        Tensor tensor(this->shape, false);
        if(this->host)
        {
            hipMemcpy(tensor.data, this->data, this->size * sizeof(scalar_t), hipMemcpyHostToDevice);
        }
        else
        {
            hipMemcpy(tensor.data, this->data, this->size * sizeof(scalar_t), hipMemcpyDeviceToDevice);
        }
        return tensor;
    }
    void to_cpu()
    {
        if(!host)
        {
            scalar_t* data_ = allocate_cpu();
            hipMemcpy(data_, data, size * sizeof(scalar_t), hipMemcpyDeviceToHost);
            hipFree(data);
            data = data_;
            host = true;
        }
    }
    void to_gpu()
    {
        if(host)
        {
            scalar_t* data_ = allocate_gpu();
            hipMemcpy(data_, data, size * sizeof(scalar_t), hipMemcpyHostToDevice);
            delete[] data;
            data = data_;
            host = false;
        }
    }
};

void relu_forward_cpu(scalar_t* in, scalar_t* out, int n)
{
    for(int i = 0; i < n; ++i)
    {
        out[i] = in[i] > 0 ? in[i] : 0;
    }
}

void relu_backward_cpu(scalar_t* out_grad, scalar_t* in, scalar_t* in_grad, int n)
{
    for(int i = 0; i < n; ++i)
    {
        in_grad[i] = in[i] > 0 ? out_grad[i] : 0;
    }
}

void sigmoid_forward_cpu(scalar_t* in, scalar_t* out, int n)
{
    for(int i = 0; i < n; ++i)
    {
        out[i] = 1.0f / (1.0f + expf(-in[i]));
    }
}

void sigmoid_backward_cpu(scalar_t* out_grad, scalar_t* in, scalar_t* in_grad, int n)
{
    for(int i = 0; i < n; ++i)
    {
        scalar_t out = 1.0f / (1.0f + expf(-in[i]));
        in_grad[i] = out_grad[i] * out * (1 - out);
    }
}

void relu_forward(Tensor* in, Tensor* out)
{
    assert(in->shape == out->shape);
    assert(in->host == out-> host);
    assert(in->size == out->size);
    if(out->host)
    {
        relu_forward_cpu(in->data, out->data, out->size);
    }
    else
    {
        relu_forward_gpu<<<CudaGetBlocks(out->size), kCudaThreadsNum>>>(in->data, out->data, out->size);
    }
}
void relu_backward(Tensor* out_grad, Tensor* in, Tensor* in_grad)
{
    assert(in_grad->shape == out_grad->shape && in_grad->shape == in->shape);
    assert(in_grad->host == out_grad->host && in_grad->host == in->host);
    assert(in_grad->size == out_grad->size && in_grad->size == in->size);
    if(in_grad->host)
    {
        relu_backward_cpu(out_grad->data, in->data, in_grad->data, in_grad->size);
    }
    else
    {
        relu_backward_gpu<<<CudaGetBlocks(in_grad->size), kCudaThreadsNum>>>(out_grad->data, in->data, in_grad->data, in_grad->size);
    }
}
void sigmoid_forward(Tensor* in, Tensor* out)
{
    assert(in->shape == out->shape);
    assert(in->host == out-> host);
    assert(in->size == out->size);
    if(out->host)
    {
        sigmoid_forward_cpu(in->data, out->data, out->size);
    }
    else
    {
        sigmoid_forward_gpu<<<CudaGetBlocks(out->size), kCudaThreadsNum>>>(in->data, out->data, out->size);
    }
}
void sigmoid_backward(Tensor* out_grad, Tensor* in, Tensor* in_grad)
{
    assert(in_grad->shape == out_grad->shape && in_grad->shape == in->shape);
    assert(in_grad->host == out_grad->host && in_grad->host == in->host);
    assert(in_grad->size == out_grad->size && in_grad->size == in->size);
    if(in_grad->host)
    {
        sigmoid_backward_cpu(out_grad->data, in->data, in_grad->data, in_grad->size);
    }
    else
    {
        sigmoid_backward_gpu<<<CudaGetBlocks(in_grad->size), kCudaThreadsNum>>>(out_grad->data, in->data, in_grad->data, in_grad->size);
    }
}

/*
TEST PART BELOW
*/

bool check(scalar_t* x, scalar_t* y, int n)
{
    for(int i = 0; i < n; i++)
    {
        if(abs(x[i] - y[i]) > 1e-6)
            return false;
    }
    return true;
}
void print(scalar_t* x, int n)
{
    for(int i = 0; i < n; i++)
    {
        std::cout<<x[i]<<' ';
    }
    std::cout<<'\n';
}

int main(int argc, char *argv[])
{
    std::vector<int> shape;
    for(int i = 2; i < argc; i++)
    {
        shape.push_back(atoi(argv[i]));
    }
    int size = 1;
    for(int dim: shape)
    {
        // std::cout<<dim<<std::endl;
        size *= dim;
    }
    Tensor input(shape), test_output(shape), test_in_grad(shape), out_grad(shape);
    // read test_data.txt
    std::ifstream data("test_data.txt");
    if(!data.is_open())
    {
        std::cout<<"Unable to open test data file";
        return -1;
    }
    else
    {
        for(int i = 0; i < size; i++)
        {
            data>>input.data[i];
        }
        for(int i = 0; i < size; i++)
        {
            data>>test_output.data[i];
        }
        for(int i = 0; i < size; i++)
        {
            data>>test_in_grad.data[i];
        }
        for(int i = 0; i < size; i++)
        {
            data>>out_grad.data[i];
        }
    }
    // simple move test
    input.to_gpu();
    hipDeviceSynchronize();
    Tensor tmp = input.gpu();
    hipDeviceSynchronize();
    tmp.to_cpu();
    input.to_cpu();
    hipDeviceSynchronize();
    if(check(tmp.data, input.data, size))
    {
        std::cout<<"Move successfully\n";
    }
    else
    {
        std::cout<<"Move failed\n";
        std::cout<<"Before:\n";
        print(input.data, size);
        std::cout<<"After:\n";
        print(tmp.data, size);
        return -1;
    }
    // move test
    // cpu test
    Tensor output_cpu(shape), in_grad_cpu(shape);
    if(strcmp(argv[1], "relu") == 0)
    {
        // forward
        relu_forward(&input, &output_cpu);
        if(check(output_cpu.data, test_output.data, size))
        {
            for(int i = 0 ; i < size; i++) {
                printf("%f ",output_cpu.data[i]);
            }
            printf("\n");
            std::cout<<"Test: relu forward on cpu    passed\n";
        }
        else
        {
            std::cout<<"Test: relu forward on cpu    failed\n";
            std::cout<<"Relu forward result on cpu:\n";
            print(output_cpu.data, size);
            std::cout<<"Correct result:\n";
            print(test_output.data, size);
            return -1;
        }
        // backward
        relu_backward(&out_grad, &input, &in_grad_cpu);
        if(check(in_grad_cpu.data, test_in_grad.data, size))
        {
            for(int i = 0 ; i < size; i++) {
                printf("%f ",in_grad_cpu.data[i]);
            }
            printf("\n");
            std::cout<<"Test: relu backward on cpu    passed\n";
        }
        else
        {
            std::cout<<"Test: relu backward on cpu    failed\n";
            std::cout<<"Relu backward result on cpu:\n";
            print(in_grad_cpu.data, size);
            std::cout<<"Correct result:\n";
            print(test_in_grad.data, size);
            return -1;
        }
    }
    else
    {
        // forward
        sigmoid_forward(&input, &output_cpu);
        if(check(output_cpu.data, test_output.data, size))
        {
            for(int i = 0 ; i < size; i++) {
                printf("%f ",output_cpu.data[i]);
            }
            printf("\n");
            std::cout<<"Test: sigmoid forward on cpu    passed\n";
        }
        else
        {
            std::cout<<"Test: sigmoid forward on cpu    failed\n";
            std::cout<<"Sigmoid forward result on cpu:\n";
            print(output_cpu.data, size);
            std::cout<<"Correct result:\n";
            print(test_output.data, size);
            return -1;
        }
        // backward
        sigmoid_backward(&out_grad, &input, &in_grad_cpu);
        if(check(in_grad_cpu.data, test_in_grad.data, size))
        {
            for(int i = 0 ; i < size; i++) {
                printf("%f ",in_grad_cpu.data[i]);
            }
            printf("\n");
            std::cout<<"Test: sigmoid backward on cpu    passed\n";
        }
        else
        {
            std::cout<<"Test: sigmoid backward on cpu    failed\n";
            std::cout<<"Sigmoid backward result on cpu:\n";
            print(in_grad_cpu.data, size);
            std::cout<<"Correct result:\n";
            print(test_in_grad.data, size);
            return -1;
        }
    }
    // gpu test
    input.to_gpu();out_grad.to_gpu();
    Tensor output_gpu(shape, false), in_grad_gpu(shape, false);
    hipDeviceSynchronize();
    if(strcmp(argv[1], "relu") == 0)
    {
        // forward
        relu_forward(&input, &output_gpu);
        output_gpu.to_cpu();
        hipDeviceSynchronize();
        if(check(output_gpu.data, test_output.data, size))
        {
            for(int i = 0 ; i < size; i++) {
                printf("%f ",output_gpu.data[i]);
            }
            printf("\n");
            std::cout<<"Test: relu forward on gpu    passed\n";
        }
        else
        {
            std::cout<<"Test: relu forward on gpu    failed\n";
            std::cout<<"Relu forward result on gpu:\n";
            print(output_gpu.data, size);
            std::cout<<"Correct result:\n";
            print(test_output.data, size);
            return -1;
        }
        // backward
        relu_backward(&out_grad, &input, &in_grad_gpu);
        in_grad_gpu.to_cpu();
        hipDeviceSynchronize();
        if(check(in_grad_gpu.data, test_in_grad.data, size))
        {
            for(int i = 0 ; i < size; i++) {
                printf("%f ",in_grad_gpu.data[i]);
            }
            printf("\n");
            std::cout<<"Test: relu backward on gpu    passed\n";
        }
        else
        {
            std::cout<<"Test: relu backward on gpu    failed\n";
            std::cout<<"Relu backward result on gpu:\n";
            print(in_grad_gpu.data, size);
            std::cout<<"Correct result:\n";
            print(test_in_grad.data, size);
            return -1;
        }
    }
    else
    {
        // forward
        sigmoid_forward(&input, &output_gpu);
        output_gpu.to_cpu();
        hipDeviceSynchronize();
        if(check(output_gpu.data, test_output.data, size))
        {
            for(int i = 0 ; i < size; i++) {
                printf("%f ",output_gpu.data[i]);
            }
            printf("\n");
            std::cout<<"Test: sigmoid forward on gpu    passed\n";
        }
        else
        {
            std::cout<<"Test: sigmoid forward on gpu    failed\n";
            std::cout<<"Sigmoid forward result on gpu:\n";
            print(output_gpu.data, size);
            std::cout<<"Correct result:\n";
            print(test_output.data, size);
            return -1;
        }
        // backward
        sigmoid_backward(&out_grad, &input, &in_grad_gpu);
        in_grad_gpu.to_cpu();
        hipDeviceSynchronize();
        if(check(in_grad_gpu.data, test_in_grad.data, size))
        {
            for(int i = 0 ; i < size; i++) {
                printf("%f ",in_grad_gpu.data[i]);
            }
            printf("\n");
            std::cout<<"Test: sigmoid backward on gpu    passed\n";
        }
        else
        {
            std::cout<<"Test: sigmoid backward on gpu    failed\n";
            std::cout<<"Sigmoid backward result on gpu:\n";
            print(in_grad_gpu.data, size);
            std::cout<<"Correct result:\n";
            print(test_in_grad.data, size);
            return -1;
        }
    }
    std::cout<<"All passed";
}