#include <vector>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <fstream>
#include <cstring>

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

const int kCudaThreadsNum = 512;//threads in block
inline int CudaGetBlocks(int N) { 
    return (N + kCudaThreadsNum - 1) / kCudaThreadsNum; 
}

float relu_cpu(float x) {
    return x > 0 ? x : 0;
}

float sigmoid_cpu(float x) {
    return 1.0f / (1.0f + expf(-x));
}

__global__ 
void relu_forward_gpu(float* input, float* output, int size) {
    CUDA_KERNEL_LOOP(i, size)
        output[i] = input[i] > 0 ? input[i] : 0;
}

__global__ 
void relu_backward_gpu(float* input, float* output, float* grad_output, float* grad_input, int size) {
    CUDA_KERNEL_LOOP(i, size) {
       grad_input[i] = input[i] > 0 ? grad_output[i] : 0;
    }
}

__global__ 
void sigmoid_forward_gpu(float* input, float* output, int size) {
    CUDA_KERNEL_LOOP(i, size) {
        float exp_val = expf(-input[i]);
        output[i] = 1.0f / (1.0f + exp_val);
    }
}

__global__ 
void sigmoid_backward_gpu(float* input, float* output, float* grad_output, float* grad_input, int size) {
    CUDA_KERNEL_LOOP(i, size) {
        float sigmoid = output[i];
        grad_input[i] = grad_output[i] * sigmoid * (1 - sigmoid);
    }
}

class Tensor {
//private:
public:
    std::vector<int> shape;
    int total_size;
    float* data;
    bool is_on_gpu;


    Tensor(const std::vector<int>& shape, bool use_gpu) : shape(shape), is_on_gpu(use_gpu) {
        total_size = 1;
        data = nullptr;
        for (auto dim : shape) {
            total_size *= dim;
        }
        if (is_on_gpu) {
            hipMalloc(&data, total_size * sizeof(float));
        } else {
            data = new float[total_size];
        }
    }

    Tensor(const std::vector<int>& shape, bool use_gpu, float* data) : shape(shape), is_on_gpu(use_gpu), data(data) {
        total_size = 1;
        for (auto dim : shape) {
            total_size *= dim;
        }
    }

    ~Tensor() {
        if(data) {
            if (is_on_gpu) {
                hipFree(data);
            } else {
                delete[] data;
            }
        }
    }

    // size
    int size() {
        return total_size;
    }

    // index
    float& operator[](int index) {
        return data[index];
    }

    // GPU to CPU
    Tensor cpu() {
        // float* cpu_data = new float[size()];
        // if (is_on_gpu) {
        //     cudaMemcpy(cpu_data, data, sizeof(float) * size(), cudaMemcpyDeviceToHost);
        // }
        // return Tensor(shape, false, cpu_data);        
        Tensor tensor(this->shape, false);
        if(this->is_on_gpu){
            hipMemcpy(tensor.data, this->data, this->size() * sizeof(float), hipMemcpyDeviceToHost);
        }
        else {
            hipMemcpy(tensor.data, this->data, this->size() * sizeof(float), hipMemcpyHostToHost);
        }
        return tensor;
    }

    // CPU to GPU
    Tensor gpu() {
        // float* gpu_data = nullptr;
        // cudaMalloc(&gpu_data, sizeof(float) * size());
        // if (!is_on_gpu) {
        //     cudaMemcpy(gpu_data, data, sizeof(float) * size(), cudaMemcpyHostToDevice);
        // }
        // return Tensor(shape, true, gpu_data);
        Tensor tensor(this->shape, true);
        if(this->is_on_gpu){
            hipMemcpy(tensor.data, this->data, this->size() * sizeof(float), hipMemcpyDeviceToDevice);
        }
        else {
            hipMemcpy(tensor.data, this->data, this->size() * sizeof(float), hipMemcpyHostToDevice);
        }
        return tensor;
    }
    void to_cpu() {
        if(is_on_gpu) {
            float* data_ = new float[size()];
            hipMemcpy(data_, data, size() * sizeof(float), hipMemcpyDeviceToHost);
            hipFree(data);
            data = data_;
            is_on_gpu = false;
        }
    }
    void to_gpu() {
        if(!is_on_gpu) {
            float* data_ = nullptr;
            hipMalloc(&data_, size() * sizeof(float));
            hipMemcpy(data_, data, size() * sizeof(float), hipMemcpyHostToDevice);
            delete[] data;
            data = data_;
            is_on_gpu = true;
        }
    }
};

void ReLUForwardCPU(Tensor* input, Tensor* output) {
    for(int i = 0; i < input->size(); i++) {
        output->data[i] = relu_cpu(input->data[i]);
    }
}

void ReLUBackwardCPU(Tensor* input, Tensor* output,Tensor* grad_output, Tensor* grad_input) {
    for(int i = 0; i < input->size(); i++) {
        grad_input->data[i] = input->data[i] > 0 ? grad_output->data[i] : 0;
    }
}

void SigmoidForwardCPU(Tensor* input, Tensor* output) {
    for(int i = 0; i < input->size(); i++) {
        output->data[i] = sigmoid_cpu(input->data[i]);
    }
}

void SigmoidBackwardCPU(Tensor* input, Tensor* output,Tensor* grad_output, Tensor* grad_input) {
    float sigmoidval;
    for(int i = 0; i < input->size(); i++) {
        sigmoidval = output->data[i];
        grad_input->data[i] = grad_output->data[i] * sigmoidval * (1 - sigmoidval);
    }
}

void ReLUForwardGPU(Tensor* input, Tensor* output) {
    relu_forward_gpu<<<CudaGetBlocks(input->size()), kCudaThreadsNum>>>(input->data, output->data, input->size());
}

void ReLUBackwardGPU(Tensor* input, Tensor* output,Tensor* grad_output, Tensor* grad_input) {
    relu_backward_gpu<<<CudaGetBlocks(input->size()), kCudaThreadsNum>>>(input->data, output->data, grad_output->data, grad_input->data, input->size());
}

void SigmoidForwardGPU(Tensor* input, Tensor* output) {
    sigmoid_forward_gpu<<<CudaGetBlocks(input->size()), kCudaThreadsNum>>>(input->data, output->data, input->size());
}

void SigmoidBackwardGPU(Tensor* input, Tensor* output,Tensor* grad_output, Tensor* grad_input) {
    sigmoid_backward_gpu<<<CudaGetBlocks(input->size()), kCudaThreadsNum>>>(input->data, output->data, grad_output->data, grad_input->data, input->size());
}


// test
int main(int argc, char *argv[])
{
    float* data_ = nullptr;
    hipError_t status = hipMalloc(&data_, sizeof(float));
    hipError_t lastError = hipGetLastError();
    if (status != hipSuccess || lastError != hipSuccess) {
        fprintf(stderr, "hipMalloc failed or last error occurred! Error: %s\n", hipGetErrorString(status));
        // 处理错误
    }
    std::vector<int> shape;
    for(int i = 2; i < argc; i++){
        shape.push_back(atoi(argv[i]));
    }
    int size = 1;
    for(int dim: shape){
        size *= dim;
    }

    // cpu test
    // Tensor input(shape, false), output_test(shape, false), grad_input_test(shape, false), grad_output_test(shape, false);
    // Tensor output_cpu(shape, false), grad_input_cpu(shape, false);

    //gpu test
    Tensor input(shape, false), output_test(shape, false), grad_input_test(shape, false), grad_output_test(shape, false);
    Tensor output_gpu(shape, true), grad_input_gpu(shape, true);

    std::ifstream data("test_data.txt");
    if(!data.is_open())
    {
        std::cout<<"Unable to open test data file";
        return -1;
    }
    else
    {
        for(int i = 0; i < size; i++) {
            data>>input.data[i];
        }
        for(int i = 0; i < size; i++) {
            data>>output_test.data[i];
        }
        for(int i = 0; i < size; i++) {
            data>>grad_input_test.data[i];
        }
        for(int i = 0; i < size; i++) {
            data>>grad_output_test.data[i];
        }
        
        //gpu
        input.to_gpu();
        output_test.to_gpu();
        grad_input_test.to_gpu();
        grad_output_test.to_gpu();
        hipDeviceSynchronize();
    }
    // cout<<size<<endl;

    // //relu_forward_cpu
    // printf("relu_forward_cpu\n");
    // ReLUForwardCPU(&input, &output_cpu);
    // for(int i = 0; i < size; i++) {
    //     printf("%f ",output_cpu.data[i]);
    // }
    // printf("\n");
    
    // //relu_backward_cpu
    // printf("relu_backward_cpu\n");
    // ReLUBackwardCPU(&input, &output_test, &grad_output_test, &grad_input_cpu);
    // for(int i = 0; i < size; i++) {
    //     printf("%f ",grad_input_cpu.data[i]);
    // }
    // printf("\n");
    
    // //sigmoid_forward_cpu
    // printf("sigmoid_forward_cpu\n");
    // SigmoidForwardCPU(&input, &output_cpu);
    // for(int i = 0; i < size; i++) {
    //     printf("%f ",output_cpu.data[i]);
    // }
    // printf("\n");

    // //sigmoid_backward_cpu
    // printf("sigmoid_backward_cpu\n");
    // SigmoidBackwardCPU(&input, &output_test, &grad_output_test, &grad_input_cpu);
    // for(int i = 0; i < size; i++) {
    //     printf("%f ",grad_input_cpu.data[i]);
    // }
    // printf("\n");



    //relu_forward_gpu
    // printf("relu_forward_gpu\n");
    // ReLUForwardGPU(&input, &output_gpu);
    // output_gpu.to_cpu();
    // cudaDeviceSynchronize();
    // for(int i = 0; i < size; i++) {
    //     printf("%f ",output_gpu.data[i]);
    // }
    // printf("\n");
    
    //relu_backward_gpu
    // printf("relu_backward_gpu\n");
    // ReLUBackwardGPU(&input, &output_test, &grad_output_test, &grad_input_gpu);
    // grad_input_gpu.to_cpu();
    // cudaDeviceSynchronize();
    // for(int i = 0; i < size; i++) {
    //     printf("%f ",grad_input_gpu.data[i]);
    // }
    // printf("\n");
    
    //sigmoid_forward_gpu
    printf("sigmoid_forward_gpu\n");
    SigmoidForwardGPU(&input, &output_gpu);
    output_gpu.to_cpu();
    hipDeviceSynchronize();
    for(int i = 0; i < size; i++) {
        printf("%f ",output_gpu.data[i]);
    }
    printf("\n");

    //sigmoid_backward_gpu
    printf("sigmoid_backward_gpu\n");
    SigmoidBackwardGPU(&input, &output_test, &grad_output_test, &grad_input_gpu);
    grad_input_gpu.to_cpu();
    hipDeviceSynchronize();
    for(int i = 0; i < size; i++) {
        printf("%f ",grad_input_gpu.data[i]);
    }
    printf("\n");
    return 0;
}